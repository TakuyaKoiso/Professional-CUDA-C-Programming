
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

int main(int argc, char ** argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
    return 0;
}
